#include "hip/hip_runtime.h"
// ========================================
// BatchACMMP.cu - Implementation with parallel disk I/O
// ========================================

#include "BatchACMMP.h"
#include <iostream>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <sstream>
#include <iomanip>

void checkCudaLimits() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    printf("Max Texture 2D: %dx%d\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
    printf("Max Texture 2D Layered: %dx%dx%d\n", 
           prop.maxTexture2DLayered[0], prop.maxTexture2DLayered[1], prop.maxTexture2DLayered[2]);
    printf("Max Surface 2D: %dx%d\n", prop.maxSurface2D[0], prop.maxSurface2D[1]);
    printf("Max Grid Size: %dx%dx%d\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    
    // Critical: texture reference limit
    printf("Max Textures per kernel: %d\n", prop.maxTexture1DLinear);
    printf("Total constant memory: %zu\n", prop.totalConstMem);
}

// ProblemGPUResources implementation
ProblemGPUResources::ProblemGPUResources() {
    for (int i = 0; i < MAX_IMAGES; ++i) {
        cuArray[i] = nullptr;
        cuDepthArray[i] = nullptr;
    }
}

void ProblemGPUResources::allocate(int max_width, int max_height, int max_images) {
    // This function is called once per resource object when the pool is initialized.
    
    // Allocate arrays for images and depths using the maximum possible dimensions.
    for (int i = 0; i < max_images; ++i) {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        CUDA_CHECK(hipMallocArray(&cuArray[i], &channelDesc, max_width, max_height));
        CUDA_CHECK(hipMallocArray(&cuDepthArray[i], &channelDesc, max_width, max_height));
    }

    // Allocate all other required device memory buffers.
    CUDA_CHECK(hipMalloc(&cameras_cuda, sizeof(Camera) * max_images));
    CUDA_CHECK(hipMalloc(&texture_objects_cuda, sizeof(cudaTextureObjects)));
    CUDA_CHECK(hipMalloc(&texture_depths_cuda, sizeof(cudaTextureObjects)));
    CUDA_CHECK(hipMalloc(&plane_hypotheses_cuda, sizeof(float4) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&scaled_plane_hypotheses_cuda, sizeof(float4) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&costs_cuda, sizeof(float) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&pre_costs_cuda, sizeof(float) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&rand_states_cuda, sizeof(hiprandState) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&selected_views_cuda, sizeof(unsigned int) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&depths_cuda, sizeof(float) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&prior_planes_cuda, sizeof(float4) * max_width * max_height));
    CUDA_CHECK(hipMalloc(&plane_masks_cuda, sizeof(unsigned int) * max_width * max_height));

    // Allocate pinned host memory for high-speed asynchronous transfers.
    CUDA_CHECK(hipHostMalloc(&planes_host_pinned, sizeof(float4) * max_width * max_height));
    CUDA_CHECK(hipHostMalloc(&costs_host_pinned, sizeof(float) * max_width * max_height));
}

void BatchACMMP::initializeResourcePool() {
    // Determine the maximum resource dimensions needed for any problem in the batch.
    // This ensures all our pooled resources are large enough.
    int max_width = 0, max_height = 0, max_images = 0;
    for (const auto& p : all_problems) {
        // A robust way to get dimensions would be to read the camera file for each problem.
        // For simplicity, we use a fixed upper bound, but reading the files is better.
        // This is a placeholder; you should replace it with actual dimension fetching logic
        // if your image sizes vary significantly.
        std::stringstream cam_path;
        cam_path << dense_folder << "/cams/" << std::setw(8) << std::setfill('0') << p.ref_image_id << "_cam.txt";
        Camera cam = ReadCamera(cam_path.str());
        
        max_width = std::max(max_width, (int)cam.width);
        max_height = std::max(max_height, (int)cam.height);
        max_images = std::max(max_images, (int)(1 + p.src_image_ids.size()));
    }
    // Clamp max_images to the maximum supported by the static array.
    max_images = std::min(max_images, MAX_IMAGES);
    
    std::cout << "[BatchACMMP] Allocating resources for max dimensions: " 
              << max_width << "x" << max_height << " with up to " << max_images << " images." << std::endl;

    streams.resize(max_concurrent_problems);
    resource_pool.resize(max_concurrent_problems);

    int prio_low=0, prio_high=0;
    CUDA_CHECK(hipDeviceGetStreamPriorityRange(&prio_low, &prio_high));
    
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        CUDA_CHECK(hipStreamCreateWithPriority(&streams[i], hipStreamNonBlocking, prio_high));

        std::unique_ptr<ProblemGPUResources> res(new ProblemGPUResources());
        res->stream_id = (int)i;
        res->stream = streams[i];

        // Allocate the GPU memory for this resource object.
        res->allocate(max_width, max_height, max_images);

        available_resources.push(res.get());
        resource_pool[i] = std::move(res);
    }
    
    // Launch GPU worker threads.
    gpu_worker_threads.reserve(max_concurrent_problems);
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        gpu_worker_threads.emplace_back(&BatchACMMP::gpuWorkerFunction, this);
    }
    
    std::cout << "[BatchACMMP] Created " << max_concurrent_problems << " GPU worker threads" << std::endl;
}


ProblemGPUResources::~ProblemGPUResources() { 
    cleanup(); 
}

// In BatchACMMP.cu, replace the entire cleanup function with this one.

void ProblemGPUResources::cleanup() {
    // Don't synchronize the stream here - it's owned by BatchACMMP
    // Just clean up the resources allocated by this object
    
    for (int i = 0; i < MAX_IMAGES; ++i) {
        if (cuArray[i]) { 
            hipFreeArray(cuArray[i]); 
            cuArray[i] = nullptr; 
        }
        if (cuDepthArray[i]) { 
            hipFreeArray(cuDepthArray[i]); 
            cuDepthArray[i] = nullptr; 
        }
    }

    // C++11 COMPATIBLE FIX: Define the lambda to take void*&
    auto safeFree = [](void*& ptr, const char* name) {
        if (ptr) {
            hipError_t err = hipFree(ptr);
            if (err != hipSuccess && err != hipErrorDeinitialized) {
                // Don't print errors during shutdown
            }
            ptr = nullptr;
        }
    };

    // C++11 COMPATIBLE FIX: Add a (void*&) cast to every call
    safeFree((void*&)cameras_cuda, "cameras_cuda");
    safeFree((void*&)texture_objects_cuda, "texture_objects_cuda");
    safeFree((void*&)texture_depths_cuda, "texture_depths_cuda");
    safeFree((void*&)plane_hypotheses_cuda, "plane_hypotheses_cuda");
    safeFree((void*&)scaled_plane_hypotheses_cuda, "scaled_plane_hypotheses_cuda");
    safeFree((void*&)costs_cuda, "costs_cuda");
    safeFree((void*&)pre_costs_cuda, "pre_costs_cuda");
    safeFree((void*&)rand_states_cuda, "rand_states_cuda");
    safeFree((void*&)selected_views_cuda, "selected_views_cuda");
    safeFree((void*&)depths_cuda, "depths_cuda");
    safeFree((void*&)prior_planes_cuda, "prior_planes_cuda");
    safeFree((void*&)plane_masks_cuda, "plane_masks_cuda");

    if (planes_host_pinned) { 
        hipHostFree(planes_host_pinned); 
        planes_host_pinned = nullptr; 
    }
    if (costs_host_pinned) { 
        hipHostFree(costs_host_pinned); 
        costs_host_pinned = nullptr; 
    }
    
    // Clear the stream reference (don't destroy it - BatchACMMP owns it)
    stream = nullptr;
}

// BatchACMMP implementation
BatchACMMP::BatchACMMP(const std::string& dense_folder_, 
                       const std::vector<Problem>& problems,
                       bool geom_consistency_,
                       bool planar_prior_,
                       bool hierarchy_,
                       bool multi_geometry_,
                       size_t mask_disk_queue_size_)
    : dense_folder(dense_folder_), all_problems(problems),
      geom_consistency(geom_consistency_), planar_prior(planar_prior_),
      hierarchy(hierarchy_), multi_geometry(multi_geometry_),mask_disk_queue_size(mask_disk_queue_size_) 
{
    // Device properties
    hipDeviceProp_t prop{};
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    checkCudaLimits();
    available_gpu_memory = getAvailableGPUMemory();
    memory_per_problem = problems.empty() ? (size_t)500 * 1024 * 1024
                                          : estimateMemoryPerProblem(problems[0]);

    // Calculate optimal GPU concurrency
    size_t usable_gpu = size_t(double(available_gpu_memory) * 0.75);
    size_t by_gpu_mem = std::max<size_t>(1, usable_gpu / memory_per_problem);
    
    // Hardware-based limits
    size_t hardware_threads = std::thread::hardware_concurrency();
    size_t by_sm = (prop.multiProcessorCount >= 20) ? 8 :
                   (prop.multiProcessorCount >= 10) ? 6 :
                   (prop.multiProcessorCount >= 5)  ? 4 : 2;
    
    max_concurrent_problems = std::min({by_gpu_mem, by_sm, size_t(12)});
    max_concurrent_problems = std::max<size_t>(1, 12);
    
    // Separate disk writer threads - optimize for disk I/O
    num_disk_writers = std::min<size_t>(4, std::max<size_t>(2, hardware_threads / 4));
    
    std::cout << "[BatchACMMP] Configuration:" << std::endl;
    std::cout << "  GPU Streams: " << max_concurrent_problems << std::endl;
    std::cout << "  Disk Writers: " << num_disk_writers << std::endl;
    std::cout << "  GPU Memory: " << (available_gpu_memory/(1024*1024)) << "MB free, "
              << (memory_per_problem/(1024*1024)) << "MB/problem" << std::endl;

    initializeResourcePool();
    initializeDiskWriters();
}

// ========================================
// Fixed BatchACMMP destructor in BatchACMMP.cu
// ========================================

BatchACMMP::~BatchACMMP() {
    // Step 1: Signal all threads to stop
    stopping_gpu_.store(true);
    stopping_disk_.store(true);
    
    // Step 2: Wake up all waiting threads
    gpu_queue_cv_.notify_all();
    disk_queue_cv_.notify_all();
    disk_queue_space_cv_.notify_all();  
    resource_cv_.notify_all();
    
    // Step 3: Join worker threads
    for (auto& t : gpu_worker_threads) {
        if (t.joinable()) {
            t.join();
        }
    }
    
    for (auto& t : disk_writer_threads) {
        if (t.joinable()) {
            t.join();
        }
    }
    
    // Step 4: Clean up GPU resources FIRST (they may reference streams)
    for (auto& res : resource_pool) {
        if (res && res->stream) {
            hipStreamSynchronize(res->stream);  // sync per stream
            res->cleanup();
        }
    }

    // Step 5: NOW destroy the streams
    for (auto& s : streams) {
        if (s) {
            hipStreamDestroy(s);
            s = nullptr;
        }
    }
        
    // Step 7: Final device synchronization
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess && err != hipErrorDeinitialized) {
        // Ignore errors during shutdown
    }
    
    std::cout << "[BatchACMMP] Shutdown complete. Peak memory: " 
              << getPeakMemoryUsage() << "MB" << std::endl;
}

size_t BatchACMMP::getSystemRAM() {
    std::ifstream meminfo("/proc/meminfo");
    std::string line;
    while (std::getline(meminfo, line)) {
        if (line.substr(0, 8) == "MemTotal") {
            std::istringstream iss(line);
            std::string key, value, unit;
            iss >> key >> value >> unit;
            return std::stoull(value) * 1024;
        }
    }
    return 16ULL * 1024 * 1024 * 1024; // Default 16GB
}

size_t BatchACMMP::estimateMemoryPerProblem(const Problem& problem) {
    std::stringstream cam_path;
    cam_path << dense_folder << "/cams/" << std::setw(8) << std::setfill('0')
             << problem.ref_image_id << "_cam.txt";
    Camera cam = ReadCamera(cam_path.str());

    const size_t W = cam.width, H = cam.height;
    const size_t N = 1 + problem.src_image_ids.size();

    size_t textures = N * W * H * (sizeof(float) + sizeof(float)); // images + depths
    size_t working = W * H * (2*sizeof(float4) + 3*sizeof(float)); // hypotheses + costs
    size_t misc = W * H * (sizeof(hiprandState) + sizeof(unsigned int));
    
    return (textures + working + misc) * 130 / 100; // 30% overhead
}

size_t BatchACMMP::getAvailableGPUMemory() {
    size_t free_mem=0, total=0;
    hipDeviceSynchronize();
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total));
    return free_mem;
}

size_t BatchACMMP::getProcessMemoryUsage() const {
    std::ifstream status("/proc/self/status");
    std::string line;
    while (std::getline(status, line)) {
        if (line.substr(0, 6) == "VmRSS:") {
            std::istringstream iss(line);
            std::string key, value, unit;
            iss >> key >> value >> unit;
            return std::stoull(value) * 1024;
        }
    }
    return 0;
}

void BatchACMMP::initializeDiskWriters() {
    // Launch disk writer threads
    disk_writer_threads.reserve(num_disk_writers);
    for (size_t i = 0; i < num_disk_writers; ++i) {
        disk_writer_threads.emplace_back(&BatchACMMP::diskWriterFunction, this);
    }
    
    std::cout << "[BatchACMMP] Created " << num_disk_writers << " disk writer threads" << std::endl;
}

ProblemGPUResources* BatchACMMP::acquireResources() {
    std::unique_lock<std::mutex> lk(resource_mutex_);
    resource_cv_.wait(lk, [&]{
        return !available_resources.empty() || stopping_gpu_.load();
    });
    if (stopping_gpu_.load()) return nullptr;
    
    auto* r = available_resources.front();
    available_resources.pop();
    return r;
}

void BatchACMMP::releaseResources(ProblemGPUResources* r) {
    if (!r) return;
    
    {
        std::lock_guard<std::mutex> lk(resource_mutex_);
        available_resources.push(r);
    }
    resource_cv_.notify_one();
}

void BatchACMMP::processAllProblems() {
    {
        std::lock_guard<std::mutex> lk(gpu_queue_mutex_);
        for (int i = 0; i < (int)all_problems.size(); ++i) {
            gpu_work_queue_.push(i);
        }
        problems_enqueued_.store((int)all_problems.size());
    }
    gpu_queue_cv_.notify_all();
    
    std::cout << "[BatchACMMP] Enqueued " << all_problems.size() << " problems" << std::endl;
    std::cout << "  GPU Processing: " << max_concurrent_problems << " parallel streams" << std::endl;
    std::cout << "  Disk Writing: " << num_disk_writers << " parallel writers" << std::endl;
}

void BatchACMMP::gpuWorkerFunction() {
    while (!stopping_gpu_.load()) {
        int idx = -1;

        // Get work from GPU queue
        {
            std::unique_lock<std::mutex> lk(gpu_queue_mutex_);
            gpu_queue_cv_.wait(lk, [&]{
                return stopping_gpu_.load() || !gpu_work_queue_.empty();
            });
            if (stopping_gpu_.load()) break;
            if (gpu_work_queue_.empty()) continue;
            
            idx = gpu_work_queue_.front();
            gpu_work_queue_.pop();
        }

        auto* res = acquireResources();
        if (!res) break;

        try {
            processProblemOnStream(idx, res);
        } catch (const std::exception& e) {
            std::cerr << "[GPU Worker] Exception processing problem " << idx << ": " << e.what() << std::endl;
        }
        
        releaseResources(res);

        int done = gpu_completed_.fetch_add(1) + 1;
        if (done % 50 == 0 || done == problems_enqueued_.load()) {
            std::cout << "[GPU Progress] " << done << "/" << problems_enqueued_.load() 
                      << " (" << (100 * done / problems_enqueued_.load()) << "%) - "
                      << "Disk pending: " << getPendingDiskWrites() << std::endl;
        }
    }
}

void BatchACMMP::diskWriterFunction() {
    while (!stopping_disk_.load()) {
        CompletedResult result;
        bool queue_was_full = false;
        
        // Get completed result from queue
        {
            std::unique_lock<std::mutex> lk(disk_queue_mutex_);
            disk_queue_cv_.wait(lk, [&]{
                return stopping_disk_.load() || !disk_write_queue_.empty();
            });
            if (stopping_disk_.load() && disk_write_queue_.empty()) break;
            if (disk_write_queue_.empty()) continue;
            
            queue_was_full = (disk_write_queue_.size() >= mask_disk_queue_size);
            result = std::move(disk_write_queue_.front());
            disk_write_queue_.pop();
        }
        
        // Signal that space is available (OUTSIDE the lock!)
        if (queue_was_full) {
            disk_queue_space_cv_.notify_all();
        }
        
        try {
            writeProblemToDisk(std::move(result));
        } catch (const std::exception& e) {
            std::cerr << "[Disk Writer] Exception writing problem: " << e.what() << std::endl;
        }
        
        int done = disk_completed_.fetch_add(1) + 1;
        if (done % 100 == 0 || done == problems_enqueued_.load()) {
            std::cout << "[Disk Progress] Saved " << done << "/" << problems_enqueued_.load() 
                      << " (" << (100 * done / problems_enqueued_.load()) << "%)" << std::endl;
        }
    }
}

void BatchACMMP::processProblemOnStream(int problem_idx, ProblemGPUResources* resources) {
    const Problem& problem = all_problems[problem_idx];
    hipStream_t stream = resources->stream;
    
    // Set the device explicitly for this thread.
    hipSetDevice(0);
    
    active_gpu_problems_.fetch_add(1);
    
    try {
        // hipStreamSynchronize(stream);
        // Ensure the stream is valid before creating ACMMP.
        hipError_t stream_check = hipStreamQuery(stream);
        if (stream_check == hipErrorInvalidHandle) {
            std::cerr << "Invalid stream for problem " << problem_idx << ", creating new stream" << std::endl;
            hipStreamCreate(&stream);
            resources->stream = stream;
        }
        
        // Process in an isolated scope to manage ACMMP's lifetime.
        {
            ACMMP acmmp;
            if (geom_consistency) acmmp.SetGeomConsistencyParams(multi_geometry);
            if (hierarchy) acmmp.SetHierarchyParams();

            // Set the stream for the ACMMP object to use for async operations.
            acmmp.SetStream(stream);
            
            // Initialize host-side data (reading images from disk).
            acmmp.InuputInitialization(dense_folder, all_problems, problem_idx);
            
            // Initialize CUDA space by copying data to the pre-allocated GPU buffers.
            acmmp.CudaSpaceInitialization(dense_folder, problem, resources);
            
            // Ensure everything is set up before running.
            hipError_t pre_run_check = hipGetLastError();
            if (pre_run_check != hipSuccess) {
                std::cerr << "Pre-run error: " << hipGetErrorString(pre_run_check) << std::endl;
                throw std::runtime_error("CUDA setup failed");
            }
            
            // Run the main algorithm, using the provided GPU resources.
            acmmp.RunPatchMatch(resources);
            
            
            // Extract results...
            const int width = acmmp.GetReferenceImageWidth();
            const int height = acmmp.GetReferenceImageHeight();

            cv::Mat_<float> depths(height, width);
            cv::Mat_<cv::Vec3f> normals(height, width);
            cv::Mat_<float> costs(height, width);

            for (int y = 0; y < height; ++y) {
                for (int x = 0; x < width; ++x) {
                    const int c = y * width + x;
                    const float4 plane_hypothesis = acmmp.GetPlaneHypothesis(c);
                    depths(y, x) = plane_hypothesis.w;
                    normals(y, x) = cv::Vec3f(plane_hypothesis.x, plane_hypothesis.y, plane_hypothesis.z);
                    costs(y, x) = acmmp.GetCost(c);
                }
            }

            // Queue the results for asynchronous disk writing with backpressure
            {
                std::unique_lock<std::mutex> lk(disk_queue_mutex_);
                // Wait if queue is full - this provides backpressure
                disk_queue_space_cv_.wait(lk, [&]{
                    return disk_write_queue_.size() < mask_disk_queue_size || stopping_disk_.load();
                });
                
                size_t queue_size = disk_write_queue_.size();
                disk_write_queue_.emplace(problem_idx, problem, 
                                         std::move(depths), std::move(normals), 
                                         std::move(costs), geom_consistency);
                
                // Warn if queue is getting large
                // if (queue_size > mask_disk_queue_size * 3 / 4) {  // 75%
                //     std::cout << "[Backpressure] Disk queue at " << queue_size << "/" 
                //               << mask_disk_queue_size << " - GPU throttled" << std::endl;
                // }
            }
            disk_queue_cv_.notify_one();
        } // ACMMP destructor is called here, freeing only its HOST memory.
        
        // Final sync after ACMMP is destroyed.
        // hipStreamSynchronize(stream);
        
    } catch (const std::exception& e) {
        std::cerr << "[Problem " << problem_idx << "] Exception: " << e.what() << std::endl;
        hipGetLastError(); // Clear any pending errors.
        throw;
    }
    
    active_gpu_problems_.fetch_sub(1);
}

void BatchACMMP::writeProblemToDisk(CompletedResult&& result) {
    // Create result folder
    std::stringstream result_path;
    result_path << dense_folder << "/ACMMP/2333_" << std::setw(8) 
                << std::setfill('0') << result.problem.ref_image_id;
    std::string result_folder = result_path.str();
    
    // Create directory (mkdir is thread-safe on most systems)
    makeDir(result_folder);
    
    // Write files
    std::string suffix = result.geom_consistency ? "/depths_geom.dmb" : "/depths.dmb";
    std::string depth_path = result_folder + suffix;
    std::string normal_path = result_folder + "/normals.dmb";
    std::string cost_path = result_folder + "/costs.dmb";
    
    writeDepthDmb(depth_path, result.depths);
    writeNormalDmb(normal_path, result.normals);
    writeDepthDmb(cost_path, result.costs);
}

void BatchACMMP::waitForGPUCompletion() {
    std::unique_lock<std::mutex> lk(gpu_queue_mutex_);
    gpu_queue_cv_.wait(lk, [&]{
        return gpu_completed_.load() >= problems_enqueued_.load();
    });
    
    for (auto& s : streams) {
        if (s) CUDA_CHECK(hipStreamSynchronize(s));
    }
    // hipDeviceSynchronize();
    
    std::cout << "[BatchACMMP] GPU processing complete!" << std::endl;
}

void BatchACMMP::waitForDiskCompletion() {
    std::unique_lock<std::mutex> lk(disk_queue_mutex_);
    disk_queue_cv_.wait(lk, [&]{
        return disk_completed_.load() >= problems_enqueued_.load();
    });
    
    std::cout << "[BatchACMMP] Disk writing complete!" << std::endl;
}

void BatchACMMP::waitForCompletion() {
    waitForGPUCompletion();
    
    size_t pending = getPendingDiskWrites();
    if (pending > 0) {
        std::cout << "[BatchACMMP] GPU complete. Flushing remaining " 
                  << pending << " results to disk..." << std::endl;
    }
    
    waitForDiskCompletion();
    
    // Verify all problems were written
    int total = problems_enqueued_.load();
    int written = disk_completed_.load();
    if (written == total) {
        std::cout << "[BatchACMMP] ✓ All " << total << " problems written successfully!" << std::endl;
    } else {
        std::cerr << "[BatchACMMP] ✗ WARNING: Only " << written << "/" << total 
                  << " problems written!" << std::endl;
    }
}

size_t BatchACMMP::getPeakMemoryUsage() const {
    std::lock_guard<std::mutex> lk(memory_mutex_);
    return peak_memory_usage_ / (1024 * 1024);
}

size_t BatchACMMP::getCurrentMemoryUsage() const {
    return getProcessMemoryUsage() / (1024 * 1024);
}

size_t BatchACMMP::getActiveGPUProblems() const {
    return active_gpu_problems_.load();
}

size_t BatchACMMP::getPendingDiskWrites() const {
    std::lock_guard<std::mutex> lk(disk_queue_mutex_);
    return disk_write_queue_.size();
}

size_t BatchACMMP::getCompletedDiskWrites() const {
    return disk_completed_.load();
}

// ======================================== 