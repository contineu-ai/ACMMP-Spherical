#include "hip/hip_runtime.h"
#include "BatchACMMP.h"
#include "ACMMP_device.cuh"

#include <iostream>
#include <algorithm>
#include <chrono>

// ---------- ProblemGPUResources impl ----------
ProblemGPUResources::ProblemGPUResources() {
    for (int i = 0; i < MAX_IMAGES; ++i) {
        cuArray[i] = nullptr;
        cuDepthArray[i] = nullptr;
    }
}

ProblemGPUResources::~ProblemGPUResources() { cleanup(); }

void ProblemGPUResources::cleanup() {
    // Free CUDA arrays
    for (int i = 0; i < MAX_IMAGES; ++i) {
        if (cuArray[i])     { hipFreeArray(cuArray[i]); cuArray[i] = nullptr; }
        if (cuDepthArray[i]){ hipFreeArray(cuDepthArray[i]); cuDepthArray[i] = nullptr; }
    }

    // Free device memory
    if (cameras_cuda)             { CUDA_CHECK(hipFree(cameras_cuda)); cameras_cuda = nullptr; }
    if (texture_objects_cuda)     { CUDA_CHECK(hipFree(texture_objects_cuda)); texture_objects_cuda = nullptr; }
    if (texture_depths_cuda)      { CUDA_CHECK(hipFree(texture_depths_cuda)); texture_depths_cuda = nullptr; }
    if (plane_hypotheses_cuda)    { CUDA_CHECK(hipFree(plane_hypotheses_cuda)); plane_hypotheses_cuda = nullptr; }
    if (scaled_plane_hypotheses_cuda){ CUDA_CHECK(hipFree(scaled_plane_hypotheses_cuda)); scaled_plane_hypotheses_cuda = nullptr; }
    if (costs_cuda)               { CUDA_CHECK(hipFree(costs_cuda)); costs_cuda = nullptr; }
    if (pre_costs_cuda)           { CUDA_CHECK(hipFree(pre_costs_cuda)); pre_costs_cuda = nullptr; }
    if (rand_states_cuda)         { CUDA_CHECK(hipFree(rand_states_cuda)); rand_states_cuda = nullptr; }
    if (selected_views_cuda)      { CUDA_CHECK(hipFree(selected_views_cuda)); selected_views_cuda = nullptr; }
    if (depths_cuda)              { CUDA_CHECK(hipFree(depths_cuda)); depths_cuda = nullptr; }
    if (prior_planes_cuda)        { CUDA_CHECK(hipFree(prior_planes_cuda)); prior_planes_cuda = nullptr; }
    if (plane_masks_cuda)         { CUDA_CHECK(hipFree(plane_masks_cuda)); plane_masks_cuda = nullptr; }

    if (planes_host_pinned) { CUDA_CHECK(hipHostFree(planes_host_pinned)); planes_host_pinned = nullptr; }
    if (costs_host_pinned)  { CUDA_CHECK(hipHostFree(costs_host_pinned));  costs_host_pinned  = nullptr; }
}

// ---------- BatchACMMP impl ----------

BatchACMMP::BatchACMMP(const std::string& dense_folder_, 
                       const std::vector<Problem>& problems,
                       bool geom_consistency_,
                       bool planar_prior_,
                       bool hierarchy_,
                       bool multi_geometry_)
    : dense_folder(dense_folder_), all_problems(problems),
      geom_consistency(geom_consistency_), planar_prior(planar_prior_),
      hierarchy(hierarchy_), multi_geometry(multi_geometry_)
{
    // Device props
    hipDeviceProp_t prop{};
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    int concurrentKernels = 0;
    CUDA_CHECK(hipDeviceGetAttribute(&concurrentKernels, hipDeviceAttributeConcurrentKernels, 0));

    available_gpu_memory = getAvailableGPUMemory();
    memory_per_problem   = problems.empty() ? (size_t)500 * 1024 * 1024
                                            : estimateMemoryPerProblem(problems[0]);

    const size_t usable = size_t(double(available_gpu_memory) * 0.8);
    size_t by_mem = std::max<size_t>(1, usable / std::max<size_t>(memory_per_problem, 1));
    size_t by_sm = (prop.multiProcessorCount > 0) ? std::max<size_t>(1, prop.multiProcessorCount / 2) : 8;
    size_t cap   = concurrentKernels ? 16 : 1;

    max_concurrent_problems = std::min<size_t>(std::min(by_mem, by_sm), cap);
    if (max_concurrent_problems == 0) max_concurrent_problems = 1;

    std::cout << "[BatchACMMP] GPU '" << prop.name << "', SMs=" << prop.multiProcessorCount
              << ", mem_free=" << (available_gpu_memory/(1024*1024)) << "MB, est/problem="
              << (memory_per_problem/(1024*1024)) << "MB, streams=" << max_concurrent_problems
              << std::endl;

    initializeResourcePool();
    initializeIOThreads();

    // Launch worker threads
    worker_threads.reserve(max_concurrent_problems);
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        worker_threads.emplace_back(&BatchACMMP::workerFunction, this);
    }
}

BatchACMMP::~BatchACMMP() {
    // Stop processing
    stopping_.store(true);
    queue_cv_.notify_all();
    resource_cv_.notify_all();

    // Stop I/O
    io_stopping_.store(true);
    io_cv_.notify_all();

    // Join all threads
    for (auto& t : worker_threads) {
        if (t.joinable()) t.join();
    }
    for (auto& t : io_threads_) {
        if (t.joinable()) t.join();
    }

    // Clean up resources
    for (auto& res : resource_pool) res.reset();
    for (auto& s : streams) {
        if (s) CUDA_CHECK(hipStreamDestroy(s));
    }
}

void BatchACMMP::initializeIOThreads(size_t num_io_threads) {
    io_threads_.reserve(num_io_threads);
    for (size_t i = 0; i < num_io_threads; ++i) {
        io_threads_.emplace_back(&BatchACMMP::ioWorkerFunction, this);
    }
}

void BatchACMMP::ioWorkerFunction() {
    while (!io_stopping_.load()) {
        ProcessedResult result;
        bool has_work = false;

        // Wait for I/O work
        {
            std::unique_lock<std::mutex> lk(io_mutex_);
            io_cv_.wait(lk, [&]{
                return io_stopping_.load() || !io_queue_.empty();
            });
            if (io_stopping_.load() && io_queue_.empty()) break;
            
            if (!io_queue_.empty()) {
                result = std::move(io_queue_.front());
                io_queue_.pop();
                has_work = true;
            }
        }

        if (has_work) {
            // Process result using callback or default file writing
            const Problem& problem = all_problems[result.problem_idx];
            
            // Convert to OpenCV format
            cv::Mat_<float> depths(result.height, result.width, 0.0f);
            cv::Mat_<cv::Vec3f> normals(result.height, result.width, cv::Vec3f(0,0,0));
            cv::Mat_<float> costs(result.height, result.width, 0.0f);

            const int W = result.width, H = result.height;
            for (int y = 0; y < H; ++y) {
                float* drow = depths.ptr<float>(y);
                cv::Vec3f* nrow = normals.ptr<cv::Vec3f>(y);
                float* crow = costs.ptr<float>(y);
                for (int x = 0; x < W; ++x) {
                    const int i = y * W + x;
                    const float4 ph = result.planes[i];
                    drow[x] = ph.w;
                    nrow[x] = cv::Vec3f(ph.x, ph.y, ph.z);
                    crow[x] = result.costs[i];
                }
            }

            // Use callback if provided, otherwise default file writing
            if (result_callback_) {
                result_callback_(result.problem_idx, depths, normals, costs);
            } else {
                // Default file writing
                std::stringstream result_path;
                result_path << dense_folder << "/ACMMP" << "/2333_" << std::setw(8) 
                           << std::setfill('0') << problem.ref_image_id;
                std::string result_folder = result_path.str();
                
                // Ensure directory exists
                create_directories_recursive(result_folder);
                
                std::string suffix = geom_consistency ? "/depths_geom.dmb" : "/depths.dmb";
                std::string depth_path = result_folder + suffix;
                std::string normal_path = result_folder + "/normals.dmb";
                std::string cost_path = result_folder + "/costs.dmb";
                
                writeDepthDmb(depth_path, depths);
                writeNormalDmb(normal_path, normals);
                writeDepthDmb(cost_path, costs);
            }

            int written = problems_written_.fetch_add(1) + 1;
            if (written % 10 == 0) {
                std::cout << "[I/O] Written " << written << "/" << problems_enqueued_.load() << " results\n";
            }
        }
    }
}

void BatchACMMP::enqueueResult(ProcessedResult&& result) {
    {
        std::lock_guard<std::mutex> lk(io_mutex_);
        io_queue_.push(std::move(result));
    }
    io_cv_.notify_one();
}

size_t BatchACMMP::estimateMemoryPerProblem(const Problem& problem) {
    // Read camera to get dims
    std::stringstream cam_path;
    cam_path << dense_folder << "/cams/" << std::setw(8) << std::setfill('0')
             << problem.ref_image_id << "_cam.txt";
    Camera cam = ReadCamera(cam_path.str());

    const size_t W = cam.width, H = cam.height;
    const size_t N = 1 + problem.src_image_ids.size();

    const size_t bytes_image   = W * H * sizeof(float);
    const size_t bytes_plane4  = W * H * sizeof(float4);
    const size_t bytes_float   = W * H * sizeof(float);

    size_t textures = N * bytes_image + N * bytes_float;
    size_t working  = 2*bytes_plane4 + 2*bytes_float + bytes_float;
    size_t misc     = W * H * (sizeof(hiprandState) + sizeof(unsigned int));

    return (textures + working + misc) + (64 * 1024 * 1024);
}

size_t BatchACMMP::getAvailableGPUMemory() {
    size_t free_mem=0, total=0;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total));
    return free_mem;
}

void BatchACMMP::initializeResourcePool() {
    streams.resize(max_concurrent_problems);
    resource_pool.resize(max_concurrent_problems);

    int prio_low=0, prio_high=0;
    CUDA_CHECK(hipDeviceGetStreamPriorityRange(&prio_low, &prio_high));
    
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        CUDA_CHECK(hipStreamCreateWithPriority(&streams[i], hipStreamNonBlocking, prio_high));

        std::unique_ptr<ProblemGPUResources> res(new ProblemGPUResources());
        res->stream_id = (int)i;
        res->stream    = streams[i];

        available_resources.push(res.get());
        resource_pool[i] = std::move(res);
    }
}

ProblemGPUResources* BatchACMMP::acquireResources() {
    std::unique_lock<std::mutex> lk(resource_mutex_);
    resource_cv_.wait(lk, [&]{
        return !available_resources.empty() || stopping_.load();
    });
    if (stopping_.load()) return nullptr;
    auto* r = available_resources.front();
    available_resources.pop();
    return r;
}

void BatchACMMP::releaseResources(ProblemGPUResources* r) {
    if (!r) return;
    {
        std::lock_guard<std::mutex> lk(resource_mutex_);
        available_resources.push(r);
    }
    resource_cv_.notify_one();
}

void BatchACMMP::processAllProblems() {
    {
        std::lock_guard<std::mutex> lk(queue_mutex_);
        for (int i = 0; i < (int)all_problems.size(); ++i) {
            problem_queue_.push(i);
        }
        problems_enqueued_.store((int)all_problems.size());
    }
    queue_cv_.notify_all();
    std::cout << "[BatchACMMP] Enqueued " << all_problems.size()
              << " problems across " << max_concurrent_problems << " streams.\n";
}

void BatchACMMP::processBatch(const std::vector<int>& idxs) {
    {
        std::lock_guard<std::mutex> lk(queue_mutex_);
        for (int i : idxs) problem_queue_.push(i);
        problems_enqueued_.fetch_add((int)idxs.size());
    }
    queue_cv_.notify_all();
}

void BatchACMMP::workerFunction() {
    while (!stopping_.load()) {
        int idx = -1;

        // Wait for work
        {
            std::unique_lock<std::mutex> lk(queue_mutex_);
            queue_cv_.wait(lk, [&]{
                return stopping_.load() || !problem_queue_.empty();
            });
            if (stopping_.load()) break;
            idx = problem_queue_.front();
            problem_queue_.pop();
        }

        auto* res = acquireResources();
        if (!res) break;

        processProblemOnStream(idx, res);
        releaseResources(res);

        int done = problems_completed_.fetch_add(1) + 1;
        if (done == problems_enqueued_.load()) {
            queue_cv_.notify_all();
        }
    }
}

void BatchACMMP::processProblemOnStream(int problem_idx, ProblemGPUResources* resources) {
    const Problem& problem = all_problems[problem_idx];
    hipStream_t stream = resources->stream;

    std::cout << "[S" << resources->stream_id << "] Problem " << problem_idx
              << " (ref " << problem.ref_image_id << ")\n";

    ACMMP acmmp;
    if (geom_consistency) acmmp.SetGeomConsistencyParams(multi_geometry);
    if (hierarchy)        acmmp.SetHierarchyParams();

    acmmp.SetStream(stream);
    acmmp.InuputInitialization(dense_folder, all_problems, problem_idx);
    acmmp.CudaSpaceInitialization(dense_folder, problem);
    acmmp.RunPatchMatch();

    // Synchronize stream for this specific problem
    CUDA_CHECK(hipStreamSynchronize(stream));

    const int width  = acmmp.GetReferenceImageWidth();
    const int height = acmmp.GetReferenceImageHeight();

    // Create result and immediately enqueue for I/O
    ProcessedResult result;
    result.problem_idx = problem_idx;
    result.width = width;
    result.height = height;
    result.planes.resize(size_t(width) * size_t(height));
    result.costs.resize(size_t(width) * size_t(height));

    // Extract results efficiently
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            const int c = y * width + x;
            result.planes[c] = acmmp.GetPlaneHypothesis(c);
            result.costs[c]  = acmmp.GetCost(c);
        }
    }

    // Immediately enqueue for I/O processing
    enqueueResult(std::move(result));
}

void BatchACMMP::waitForCompletion() {
    // Wait for processing to complete
    std::unique_lock<std::mutex> lk(queue_mutex_);
    queue_cv_.wait(lk, [&]{
        return problems_completed_.load() >= problems_enqueued_.load();
    });
    
    // Wait for I/O to complete
    while (problems_written_.load() < problems_enqueued_.load()) {
        std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }
    
    // Final stream sync for safety
    for (auto& s : streams) CUDA_CHECK(hipStreamSynchronize(s));
    
    std::cout << "[BatchACMMP] All " << problems_enqueued_.load() 
              << " problems processed and written.\n";
}