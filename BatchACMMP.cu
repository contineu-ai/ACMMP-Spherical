#include "hip/hip_runtime.h"
#include "BatchACMMP.h"
#include "ACMMP_device.cuh"

#include <iostream>
#include <algorithm>
#include <chrono>

// ---------- ProblemGPUResources impl ----------
ProblemGPUResources::ProblemGPUResources() {
    for (int i = 0; i < MAX_IMAGES; ++i) {
        cuArray[i] = nullptr;
        cuDepthArray[i] = nullptr;
    }
}

ProblemGPUResources::~ProblemGPUResources() { cleanup(); }

void ProblemGPUResources::cleanup() {
    // Free CUDA arrays
    for (int i = 0; i < MAX_IMAGES; ++i) {
        if (cuArray[i])     { hipFreeArray(cuArray[i]); cuArray[i] = nullptr; }
        if (cuDepthArray[i]){ hipFreeArray(cuDepthArray[i]); cuDepthArray[i] = nullptr; }
    }

    // Free device memory
    if (cameras_cuda)             { CUDA_CHECK(hipFree(cameras_cuda)); cameras_cuda = nullptr; }
    if (texture_objects_cuda)     { CUDA_CHECK(hipFree(texture_objects_cuda)); texture_objects_cuda = nullptr; }
    if (texture_depths_cuda)      { CUDA_CHECK(hipFree(texture_depths_cuda)); texture_depths_cuda = nullptr; }
    if (plane_hypotheses_cuda)    { CUDA_CHECK(hipFree(plane_hypotheses_cuda)); plane_hypotheses_cuda = nullptr; }
    if (scaled_plane_hypotheses_cuda){ CUDA_CHECK(hipFree(scaled_plane_hypotheses_cuda)); scaled_plane_hypotheses_cuda = nullptr; }
    if (costs_cuda)               { CUDA_CHECK(hipFree(costs_cuda)); costs_cuda = nullptr; }
    if (pre_costs_cuda)           { CUDA_CHECK(hipFree(pre_costs_cuda)); pre_costs_cuda = nullptr; }
    if (rand_states_cuda)         { CUDA_CHECK(hipFree(rand_states_cuda)); rand_states_cuda = nullptr; }
    if (selected_views_cuda)      { CUDA_CHECK(hipFree(selected_views_cuda)); selected_views_cuda = nullptr; }
    if (depths_cuda)              { CUDA_CHECK(hipFree(depths_cuda)); depths_cuda = nullptr; }
    if (prior_planes_cuda)        { CUDA_CHECK(hipFree(prior_planes_cuda)); prior_planes_cuda = nullptr; }
    if (plane_masks_cuda)         { CUDA_CHECK(hipFree(plane_masks_cuda)); plane_masks_cuda = nullptr; }

    if (planes_host_pinned) { CUDA_CHECK(hipHostFree(planes_host_pinned)); planes_host_pinned = nullptr; }
    if (costs_host_pinned)  { CUDA_CHECK(hipHostFree(costs_host_pinned));  costs_host_pinned  = nullptr; }
}

// ---------- BatchACMMP impl ----------

BatchACMMP::BatchACMMP(const std::string& dense_folder_, 
                       const std::vector<Problem>& problems,
                       bool geom_consistency_,
                       bool planar_prior_,
                       bool hierarchy_,
                       bool multi_geometry_)
    : dense_folder(dense_folder_), all_problems(problems),
      geom_consistency(geom_consistency_), planar_prior(planar_prior_),
      hierarchy(hierarchy_), multi_geometry(multi_geometry_)
{
    // Device props
    hipDeviceProp_t prop{};
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    int concurrentKernels = 0;
    CUDA_CHECK(hipDeviceGetAttribute(&concurrentKernels, hipDeviceAttributeConcurrentKernels, 0));

    available_gpu_memory = getAvailableGPUMemory();
    memory_per_problem   = problems.empty() ? (size_t)500 * 1024 * 1024
                                            : estimateMemoryPerProblem(problems[0]);

    const size_t usable = size_t(double(available_gpu_memory) * 0.8);
    size_t by_mem = std::max<size_t>(1, usable / std::max<size_t>(memory_per_problem, 1));
    size_t by_sm = (prop.multiProcessorCount > 0) ? std::max<size_t>(1, prop.multiProcessorCount / 2) : 8;
    size_t cap   = concurrentKernels ? 16 : 1;

    max_concurrent_problems = std::min<size_t>(std::min(by_mem, by_sm), cap);
    if (max_concurrent_problems == 0) max_concurrent_problems = 1;

    std::cout << "[BatchACMMP] GPU '" << prop.name << "', SMs=" << prop.multiProcessorCount
              << ", mem_free=" << (available_gpu_memory/(1024*1024)) << "MB, est/problem="
              << (memory_per_problem/(1024*1024)) << "MB, streams=" << max_concurrent_problems
              << std::endl;

    initializeResourcePool();
    initializeIOThreads();

    // Launch worker threads
    worker_threads.reserve(max_concurrent_problems);
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        worker_threads.emplace_back(&BatchACMMP::workerFunction, this);
    }
}

BatchACMMP::~BatchACMMP() {
    // Stop processing
    stopping_.store(true);
    queue_cv_.notify_all();
    resource_cv_.notify_all();

    // Stop I/O
    io_stopping_.store(true);
    io_cv_.notify_all();

    // Join all threads
    for (auto& t : worker_threads) {
        if (t.joinable()) t.join();
    }
    for (auto& t : io_threads_) {
        if (t.joinable()) t.join();
    }

    // Clean up resources
    for (auto& res : resource_pool) res.reset();
    for (auto& s : streams) {
        if (s) CUDA_CHECK(hipStreamDestroy(s));
    }
}

void BatchACMMP::initializeIOThreads(size_t num_io_threads) {
    io_threads_.reserve(num_io_threads);
    for (size_t i = 0; i < num_io_threads; ++i) {
        io_threads_.emplace_back(&BatchACMMP::ioWorkerFunction, this);
    }
}

void BatchACMMP::ioWorkerFunction() {
    while (!io_stopping_.load()) {
        ProcessedResult result;
        bool has_work = false;

        // Wait for I/O work
        {
            std::unique_lock<std::mutex> lk(io_mutex_);
            io_cv_.wait(lk, [&]{
                return io_stopping_.load() || !io_queue_.empty();
            });
            if (io_stopping_.load() && io_queue_.empty()) break;
            
            if (!io_queue_.empty()) {
                // CRITICAL: Move the entire result to avoid shared references
                result = std::move(io_queue_.front());
                io_queue_.pop();
                has_work = true;
            }
        }

        if (has_work) {
            try {
                const Problem& problem = all_problems[result.problem_idx];
                
                // Ensure dimensions are valid
                if (result.width <= 0 || result.height <= 0) {
                    std::cerr << "[I/O] Invalid dimensions for problem " << result.problem_idx 
                              << ": " << result.width << "x" << result.height << std::endl;
                    continue;
                }
                
                const int W = result.width, H = result.height;
                const size_t expected_size = size_t(W) * size_t(H);
                
                // Validate data sizes
                if (result.planes.size() != expected_size || result.costs.size() != expected_size) {
                    std::cerr << "[I/O] Data size mismatch for problem " << result.problem_idx 
                              << ": expected " << expected_size 
                              << ", got planes=" << result.planes.size() 
                              << ", costs=" << result.costs.size() << std::endl;
                    continue;
                }
                
                // Create output matrices with proper initialization
                cv::Mat_<float> depths(H, W);
                cv::Mat_<cv::Vec3f> normals(H, W);
                cv::Mat_<float> costs(H, W);
                
                // Safe parallel copy with bounds checking
                #pragma omp parallel for schedule(static) collapse(2)
                for (int y = 0; y < H; ++y) {
                    for (int x = 0; x < W; ++x) {
                        const int i = y * W + x;
                        if (i < expected_size) {  // Extra safety check
                            const float4& ph = result.planes[i];
                            depths(y, x) = ph.w;
                            normals(y, x) = cv::Vec3f(ph.x, ph.y, ph.z);
                            costs(y, x) = result.costs[i];
                        }
                    }
                }
                
                // Use callback if provided, otherwise default file writing
                if (result_callback_) {
                    result_callback_(result.problem_idx, depths, normals, costs);
                } else {
                    // Default file writing with error handling
                    std::stringstream result_path;
                    result_path << dense_folder << "/ACMMP" << "/2333_" << std::setw(8) 
                               << std::setfill('0') << problem.ref_image_id;
                    std::string result_folder = result_path.str();
                    
                    // Ensure directory exists
                    create_directories_recursive(result_folder);
                    
                    std::string suffix = geom_consistency ? "/depths_geom.dmb" : "/depths.dmb";
                    std::string depth_path = result_folder + suffix;
                    std::string normal_path = result_folder + "/normals.dmb";
                    std::string cost_path = result_folder + "/costs.dmb";
                    
                    // Write with error checking
                    if (writeDepthDmb(depth_path, depths) != 0) {
                        std::cerr << "[I/O] Failed to write depth for problem " << result.problem_idx << std::endl;
                    }
                    if (writeNormalDmb(normal_path, normals) != 0) {
                        std::cerr << "[I/O] Failed to write normals for problem " << result.problem_idx << std::endl;
                    }
                    if (writeDepthDmb(cost_path, costs) != 0) {
                        std::cerr << "[I/O] Failed to write costs for problem " << result.problem_idx << std::endl;
                    }
                }
                
                int written = problems_written_.fetch_add(1) + 1;
                if (written % 10 == 0) {
                    std::cout << "[I/O] Written " << written << "/" << problems_enqueued_.load() << " results\n";
                }
                
            } catch (const std::exception& e) {
                std::cerr << "[I/O] Exception processing result " << result.problem_idx 
                          << ": " << e.what() << std::endl;
            } catch (...) {
                std::cerr << "[I/O] Unknown exception processing result " << result.problem_idx << std::endl;
            }
        }
    }
}
void BatchACMMP::enqueueResult(ProcessedResult&& result) {
    {
        std::lock_guard<std::mutex> lk(io_mutex_);
        io_queue_.push(std::move(result));
    }
    io_cv_.notify_one();
}

size_t BatchACMMP::estimateMemoryPerProblem(const Problem& problem) {
    // Read camera to get dims
    std::stringstream cam_path;
    cam_path << dense_folder << "/cams/" << std::setw(8) << std::setfill('0')
             << problem.ref_image_id << "_cam.txt";
    Camera cam = ReadCamera(cam_path.str());

    const size_t W = cam.width, H = cam.height;
    const size_t N = 1 + problem.src_image_ids.size();

    const size_t bytes_image   = W * H * sizeof(float);
    const size_t bytes_plane4  = W * H * sizeof(float4);
    const size_t bytes_float   = W * H * sizeof(float);

    size_t textures = N * bytes_image + N * bytes_float;
    size_t working  = 2*bytes_plane4 + 2*bytes_float + bytes_float;
    size_t misc     = W * H * (sizeof(hiprandState) + sizeof(unsigned int));

    return (textures + working + misc) + (64 * 1024 * 1024);
}

size_t BatchACMMP::getAvailableGPUMemory() {
    size_t free_mem=0, total=0;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total));
    return free_mem;
}

void BatchACMMP::initializeResourcePool() {
    streams.resize(max_concurrent_problems);
    resource_pool.resize(max_concurrent_problems);

    int prio_low=0, prio_high=0;
    CUDA_CHECK(hipDeviceGetStreamPriorityRange(&prio_low, &prio_high));
    
    for (size_t i = 0; i < max_concurrent_problems; ++i) {
        CUDA_CHECK(hipStreamCreateWithPriority(&streams[i], hipStreamNonBlocking, prio_high));

        std::unique_ptr<ProblemGPUResources> res(new ProblemGPUResources());
        res->stream_id = (int)i;
        res->stream    = streams[i];

        available_resources.push(res.get());
        resource_pool[i] = std::move(res);
    }
}

ProblemGPUResources* BatchACMMP::acquireResources() {
    std::unique_lock<std::mutex> lk(resource_mutex_);
    resource_cv_.wait(lk, [&]{
        return !available_resources.empty() || stopping_.load();
    });
    if (stopping_.load()) return nullptr;
    
    auto* r = available_resources.front();
    available_resources.pop();
    
    // CRITICAL: Ensure previous operations on this resource are complete
    if (r && r->stream) {
        CUDA_CHECK(hipStreamSynchronize(r->stream));
    }
    
    return r;
}

void BatchACMMP::releaseResources(ProblemGPUResources* r) {
    if (!r) return;
    
    // Ensure all operations on this stream are complete
    if (r->stream) {
        CUDA_CHECK(hipStreamSynchronize(r->stream));
    }
    
    {
        std::lock_guard<std::mutex> lk(resource_mutex_);
        available_resources.push(r);
    }
    resource_cv_.notify_one();
}


void BatchACMMP::processAllProblems() {
    {
        std::lock_guard<std::mutex> lk(queue_mutex_);
        for (int i = 0; i < (int)all_problems.size(); ++i) {
            problem_queue_.push(i);
        }
        problems_enqueued_.store((int)all_problems.size());
    }
    queue_cv_.notify_all();
    std::cout << "[BatchACMMP] Enqueued " << all_problems.size()
              << " problems across " << max_concurrent_problems << " streams.\n";
}

void BatchACMMP::processBatch(const std::vector<int>& idxs) {
    {
        std::lock_guard<std::mutex> lk(queue_mutex_);
        for (int i : idxs) problem_queue_.push(i);
        problems_enqueued_.fetch_add((int)idxs.size());
    }
    queue_cv_.notify_all();
}

void BatchACMMP::workerFunction() {
    while (!stopping_.load()) {
        int idx = -1;

        // Wait for work
        {
            std::unique_lock<std::mutex> lk(queue_mutex_);
            queue_cv_.wait(lk, [&]{
                return stopping_.load() || !problem_queue_.empty();
            });
            if (stopping_.load()) break;
            idx = problem_queue_.front();
            problem_queue_.pop();
        }

        auto* res = acquireResources();
        if (!res) break;

        processProblemOnStream(idx, res);
        releaseResources(res);

        int done = problems_completed_.fetch_add(1) + 1;
        if (done == problems_enqueued_.load()) {
            queue_cv_.notify_all();
        }
    }
}

void BatchACMMP::processProblemOnStream(int problem_idx, ProblemGPUResources* resources) {
    const Problem& problem = all_problems[problem_idx];
    hipStream_t stream = resources->stream;

    std::cout << "[S" << resources->stream_id << "] Problem " << problem_idx
              << " (ref " << problem.ref_image_id << ")\n";

    // Use heap allocation to control ACMMP lifetime
    std::unique_ptr<ACMMP> acmmp(new ACMMP());
    
    if (geom_consistency) acmmp->SetGeomConsistencyParams(multi_geometry);
    if (hierarchy)        acmmp->SetHierarchyParams();

    acmmp->SetStream(stream);
    acmmp->InuputInitialization(dense_folder, all_problems, problem_idx);
    acmmp->CudaSpaceInitialization(dense_folder, problem);
    acmmp->RunPatchMatch();

    // CRITICAL: Full synchronization before accessing results
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipDeviceSynchronize()); // Extra safety
    
    const int width  = acmmp->GetReferenceImageWidth();
    const int height = acmmp->GetReferenceImageHeight();

    // Create result with pre-allocated memory
    ProcessedResult result;
    result.problem_idx = problem_idx;
    result.width = width;
    result.height = height;
    
    const size_t total_pixels = size_t(width) * size_t(height);
    result.planes.resize(total_pixels);
    result.costs.resize(total_pixels);

    // CRITICAL: Copy all data while ACMMP is still alive
    // Use temporary buffers to ensure complete copy
    std::vector<float4> temp_planes(total_pixels);
    std::vector<float> temp_costs(total_pixels);
    
    // Batch copy for better performance
    #pragma omp parallel for schedule(static)
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            const int c = y * width + x;
            temp_planes[c] = acmmp->GetPlaneHypothesis(c);
            temp_costs[c] = acmmp->GetCost(c);
        }
    }
    
    // Now safe to move data to result
    result.planes = std::move(temp_planes);
    result.costs = std::move(temp_costs);
    
    // Ensure all GPU operations are complete before destroying ACMMP
    CUDA_CHECK(hipStreamSynchronize(stream));
    
    // Explicitly destroy ACMMP before enqueueing result
    acmmp.reset();
    
    // Now safe to enqueue result for I/O
    enqueueResult(std::move(result));
}

void BatchACMMP::waitForCompletion() {
    // Wait for processing to complete
    std::unique_lock<std::mutex> lk(queue_mutex_);
    queue_cv_.wait(lk, [&]{
        return problems_completed_.load() >= problems_enqueued_.load();
    });
    
    // Wait for I/O to complete
    while (problems_written_.load() < problems_enqueued_.load()) {
        std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }
    
    // Final stream sync for safety
    for (auto& s : streams) CUDA_CHECK(hipStreamSynchronize(s));
    
    std::cout << "[BatchACMMP] All " << problems_enqueued_.load() 
              << " problems processed and written.\n";
}